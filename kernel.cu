
#include <hip/hip_runtime.h>
#include <iostream>
#define WIDTH 1920

__constant__ unsigned int c_palette[16];

__device__ int diverge(double cx,double cy, int max_iters){
    int iter=0;
    double vx=cx;
    double vy=cy;
    double tx,ty;

    while (iter<max_iters && (vx*vx+vy*vy)<4){
        tx=vx*vx-vy*vy+cx;
        ty=2*vx*vy+cy;

        vx=tx;
        vy=ty;

        iter++;
    }
    if(iter>0 && iter<max_iters){
        return c_palette[iter%16];
    }

    return 0x000000;
}

__global__ void mandelbrot_kernel(unsigned  int* buffer, double x_start, double x_end,
                                  double y_start, double y_end,
                                  double dx, double dy,
                                  int max_iters){

    int id= blockDim.x * blockIdx.x + threadIdx.x;

    int i=id % WIDTH;
    int j=id / WIDTH;

    //cx+cxi numero complejo
    double cx=x_start+i*dx;
    double cy=y_end-j*dy;
    int color=diverge(cx,cy, max_iters);

    buffer[id]=color;
}

//Exportar
extern "C" void setPalette (unsigned int* h_palette){
    hipMemcpyToSymbol(HIP_SYMBOL(c_palette), h_palette, 16*sizeof(unsigned  int));
}

extern "C" void invoke_mandelbrot_kernel(
        int block_in_grid, int threads_per_block,
        unsigned  int* buffer, double x_start, double x_end,
        double y_start, double y_end,
        double dx, double dy,
        int max_iter){

    mandelbrot_kernel<<<block_in_grid, threads_per_block>>>(
            buffer, x_start, x_end, y_start, y_end,
            dx,dy,
            max_iter
    );
}

